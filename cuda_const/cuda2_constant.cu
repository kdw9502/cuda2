#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

#include <assert.h>

/******************************
Device: GTX 1070
===============================
N_ELEMS: 2^21
Cutoff count: 1,000

******************************/
#define ELEM_PER_POINT				(1 << 7)
#define N_ELEM_LOG					(22)

#define N_ELEMS						(1 << N_ELEM_LOG)
#define N_POINTS					(N_ELEMS / ELEM_PER_POINT)

#define ARRAY_2D_WIDTH				1024
#define ARRAY_2D_HEIGHT				(N_POINTS/ARRAY_2D_WIDTH)
#define BLOCK_WIDTH					1025
#define BLOCK_HEIGHT				1

//#define MAX_SHARED_MEM_PER_BLOCK	(3 << 14)	// from GTX 680
//#define MAX_SHARED_MEM_PER_SM		(3 << 14)	// same with 'per block'

//#define SHARED_AOS_BLOCK_HEIGHT		((ELEM_PER_POINT >> 4) ? 3 : BLOCK_HEIGHT)
//#define SHARED_AOS_BLOCK_WIDTH		((ELEM_PER_POINT >> 4) ? ((MAX_SHARED_MEM_PER_BLOCK / (ELEM_PER_POINT * sizeof(float)) ) / SHARED_AOS_BLOCK_HEIGHT) : BLOCK_WIDTH)

__int64 start, freq, end;
#define CHECK_TIME_START QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start)
#define CHECK_TIME_END(a) QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f))

#define IN
#define INOUT
#define OUT


typedef struct {
	float elem[ ELEM_PER_POINT ];
} POINT_ELEMENT;

typedef struct {
	float *elem[ ELEM_PER_POINT ];
} POINTS_SOA;

__constant__ float constantBuffer[ 1000 ];
//extern __shared__ float sharedBuffer[ ];

__global__ void TransformAOSKernel( INOUT POINT_ELEMENT *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	float tmp;
	int i,j;
	for (j = 2; j <= m; j++) {
		tmp = 1.0f / (float)j;
		for(i=0;i<ELEM_PER_POINT;i++){
			A[id].elem[i] += tmp*A[id].elem[i];
		}
	}

}

__global__ void TransformSOAKernel( INOUT POINTS_SOA A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
		float tmp;
	int i,j;
	for(i=0;i<ELEM_PER_POINT;i++){
		for (j = 2; j <= m; j++) {
			tmp = 1.0f / (float)j;
		
			A.elem[i][id] += tmp*A.elem[i][id];
		}
	}

}

__global__ void TransformAOSwithConstantMemKernel( INOUT POINT_ELEMENT *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	int i,j;

	for ( i = 0; i < ELEM_PER_POINT; ++i ) {
		for ( j = 2; j <= m; j++ ) {
			A[id].elem[ i ] += constantBuffer[j-1]*A[id].elem[ i ];
		}
	}
	
}

__global__ void TransformSOAwithConstantMemKernel( INOUT POINTS_SOA A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	int i, j;

	for ( i = 0; i < ELEM_PER_POINT; ++i ) {
		for ( j = 2; j <= m; j++ ) {
			A.elem[ i ][ id ] += constantBuffer[j-1]*A.elem[ i ][ id ];
		}
	}
	
}

void transform_points_AOS( INOUT POINT_ELEMENT *p_AOS, IN int n_points, IN int m )
{
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	

//.............
	POINT_ELEMENT *d_pAOS;
	size_t size = N_POINTS * sizeof(POINT_ELEMENT);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid(ARRAY_2D_WIDTH /dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);

		hipEventRecord( start, 0 );
	TransformAOSKernel <<< dimGrid, dimBlock >>>(d_pAOS, m);
		hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);


	//hipDeviceSynchronize(); //It may stall the GPU pipeline.

	//
//.............



	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("*** GPU1 - E:%d N:%d B:(%d,%d) M:global,AOS : GPU Time taken = %.3fms\n", ELEM_PER_POINT, N_ELEM_LOG, BLOCK_HEIGHT, BLOCK_WIDTH, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
}

void transform_points_SOA( INOUT POINTS_SOA p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	

//.............
	POINTS_SOA d_pSOA;
	size_t size = N_POINTS * sizeof(float);
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid(ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);
	hipEventRecord( start, 0 );
	TransformSOAKernel <<< dimGrid, dimBlock >>> (d_pSOA, m);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);

	}
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipFree(d_pSOA.elem[i]);
	}
	//hipDeviceSynchronize(); //It may stall the GPU pipeline.

	//
//.............



	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("*** GPU2 - E:%d N:%d B:(%d,%d) M:global,SOA : GPU Time taken = %.3fms\n", ELEM_PER_POINT, N_ELEM_LOG, BLOCK_HEIGHT, BLOCK_WIDTH, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

}

void transform_points_AOS_with_constant( INOUT POINT_ELEMENT *p_AOS, IN int n_points, IN int m )
{	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	


//.............
	POINT_ELEMENT *d_pAOS;
	size_t size = N_POINTS * sizeof(POINT_ELEMENT);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid(ARRAY_2D_WIDTH /dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);
		hipEventRecord( start, 0 );
	
	TransformAOSwithConstantMemKernel <<< dimGrid, dimBlock >>>(d_pAOS, m);
		hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

		//hipDeviceSynchronize(); //It may stall the GPU pipeline.

	//
//.............



	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("*** GPU3 - E:%d N:%d B:(%d,%d) M:constant,AOS : GPU Time taken = %.3fms\n", ELEM_PER_POINT, N_ELEM_LOG, BLOCK_HEIGHT, BLOCK_WIDTH, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

}

void transform_points_SOA_with_constant( INOUT POINTS_SOA p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	

//.............
	POINTS_SOA d_pSOA;
	size_t size = N_POINTS * sizeof(float);
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid(ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);
		hipEventRecord( start, 0 );
	TransformSOAwithConstantMemKernel <<< dimGrid, dimBlock >>> (d_pSOA, m);
		hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
		hipFree(d_pSOA.elem[i]);
	}

		//hipDeviceSynchronize(); //It may stall the GPU pipeline.

	//
//.............



	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("*** GPU4 - E:%d N:%d B:(%d,%d) M:constant,SOA : GPU Time taken = %.3fms\n", ELEM_PER_POINT, N_ELEM_LOG, BLOCK_HEIGHT, BLOCK_WIDTH, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}


void generate_point_data( OUT POINT_ELEMENT *p_AOS, OUT POINTS_SOA *p_SOA, IN int n )
{
	int i;

	srand( ( unsigned int )time( NULL ) );

	for( i = 0; i < n; i++ )
	{
		for( int j = 0; j < ELEM_PER_POINT; ++j )
		{
			p_AOS[ i ].elem[ j ] = p_SOA->elem[ j ][ i ] = 100.0f*( ( float )rand() ) / RAND_MAX;
		}
	}
}

void generate_constant_data( IN int m )
{
	float *p_constant = new float[ m ];

	p_constant[ 0 ] = 0; // not used
	for( int i = 2; i <= m; ++i )
	{
		p_constant[ i-1 ] = 1.0f / ( float )i;
	}

	hipMemcpyToSymbol(HIP_SYMBOL( constantBuffer), p_constant, sizeof( float )* m );

	delete[] p_constant;
}

int main(void){

	float compute_time;
	int n_points, cutoff;
	POINT_ELEMENT *Points_AOS;
	POINTS_SOA Points_SOA;

	printf("ELEMENT PER POINT = %d\n\n", ELEM_PER_POINT);

	n_points = N_POINTS;
	cutoff = 1000;
	Points_AOS = new POINT_ELEMENT[N_POINTS];
	for( int i = 0; i < ELEM_PER_POINT; ++i )
	{
		Points_SOA.elem[ i ] = new float[ N_POINTS ];
	}
	generate_point_data(Points_AOS, &Points_SOA, n_points);
	generate_constant_data(cutoff);

	transform_points_AOS(Points_AOS, n_points, cutoff);

	transform_points_SOA(Points_SOA, n_points, cutoff);


	printf("\n");
	printf("--- AOS.10.x = %e, SOA.10.x = %e\n", Points_AOS[10].elem[0], Points_SOA.elem[0][10]);
	printf("--- AOS.20.y = %e, SOA.20.y = %e\n", Points_AOS[20].elem[1], Points_SOA.elem[1][20]);
	printf("\n");

	printf("\n///////////////////Second round///////////////////////\n\n");
	generate_point_data(Points_AOS, &Points_SOA, n_points);
	generate_constant_data(cutoff);
	transform_points_AOS(Points_AOS, n_points, cutoff);

	transform_points_SOA(Points_SOA, n_points, cutoff);

	printf("\n");
	printf("--- AOS.10.x = %e, SOA.10.x = %e\n", Points_AOS[10].elem[0], Points_SOA.elem[0][10]);
	printf("--- AOS.20.y = %e, SOA.20.y = %e\n", Points_AOS[20].elem[1], Points_SOA.elem[1][20]);
	printf("\n");

	printf( "\n///////////////////Constant first round///////////////////////\n\n" );
	generate_point_data(Points_AOS, &Points_SOA, n_points);
	generate_constant_data(cutoff);
	transform_points_AOS_with_constant(Points_AOS, n_points, cutoff);
	transform_points_SOA_with_constant(Points_SOA, n_points, cutoff);
	printf( "\n" );
	printf( "--- AOS.10.x = %e, SOA.10.x = %e\n", Points_AOS[ 10 ].elem[ 0 ], Points_SOA.elem[ 0 ][ 10 ] );
	printf( "--- AOS.20.y = %e, SOA.20.y = %e\n", Points_AOS[ 20 ].elem[ 1 ], Points_SOA.elem[ 1 ][ 20 ] );
	printf( "\n" );

	printf( "\n///////////////////Constant second round///////////////////////\n\n" );
	generate_point_data(Points_AOS, &Points_SOA, n_points);
	generate_constant_data(cutoff);
	transform_points_AOS_with_constant(Points_AOS, n_points, cutoff);
	transform_points_SOA_with_constant(Points_SOA, n_points, cutoff);
	
	printf( "\n" );
	printf( "--- AOS.10.x = %e, SOA.10.x = %e\n", Points_AOS[ 10 ].elem[ 0 ], Points_SOA.elem[ 0 ][ 10 ] );
	printf( "--- AOS.20.y = %e, SOA.20.y = %e\n", Points_AOS[ 20 ].elem[ 1 ], Points_SOA.elem[ 1 ][ 20 ] );
	printf( "\n" );


	return 0;
}